#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256

__device__ void merge(int *arr, int *temp, int l, int m, int r)
{
    int i = l, j = m + 1, k = l;

    while (i <= m && j <= r)
    {
        if (arr[i] <= arr[j])
            temp[k++] = arr[i++];
        else
            temp[k++] = arr[j++];
    }

    while (i <= m)
        temp[k++] = arr[i++];
    while (j <= r)
        temp[k++] = arr[j++];

    for (i = l; i <= r; i++)
        arr[i] = temp[i];
}
__global__ void mergeSort(int *arr, int *temp, int n)
{
    extern __shared__ int shared_mem[];

    int tid = threadIdx.x;
    int block_size = blockDim.x;
    int block_start = blockIdx.x * block_size;

    if (tid < block_size && block_start + tid < n)
    {
        shared_mem[tid] = arr[block_start + tid];
    }
    __syncthreads();

    for (int curr_size = 1; curr_size < block_size; curr_size *= 2)
    {
        for (int left_start = 0; left_start < block_size; left_start += 2 * curr_size)
        {
            int mid = left_start + curr_size - 1;
            int right_end = min(left_start + 2 * curr_size - 1, block_size - 1);

            if (mid < right_end)
            {
                merge(shared_mem, temp, left_start, mid, right_end);
            }
        }
        __syncthreads();
    }

    if (tid < block_size && block_start + tid < n)
    {
        arr[block_start + tid] = shared_mem[tid];
    }
}
void merge2(int *arr, int *temp, int l, int m, int r)
{
    int i = l, j = m + 1, k = l;

    while (i <= m && j <= r)
    {
        if (arr[i] <= arr[j])
            temp[k++] = arr[i++];
        else
            temp[k++] = arr[j++];
    }

    while (i <= m)
        temp[k++] = arr[i++];
    while (j <= r)
        temp[k++] = arr[j++];

    for (i = l; i <= r; i++)
        arr[i] = temp[i];
}
void mergeSortedBlocks(int *arr, int n, int block_size)
{
    int *temp = (int *)malloc(n * sizeof(int));

    for (int size = block_size; size < n; size *= 2)
    {
        for (int left_start = 0; left_start < n; left_start += size * 2)
        {
            int mid = left_start + size - 1;
            int right_end = min(left_start + size * 2 - 1, n - 1);

            if (mid < right_end)
            {
                merge2(arr, temp, left_start, mid, right_end);
            }
        }
    }

    free(temp);
}

int main(int argc, char *argv[])
{
    if (argc != 2)
    {
        printf("Usage: %s <array_size>\n", argv[0]);
        return 1;
    }

    int n = atoi(argv[1]);
    if (n <= 0)
    {
        printf("Invalid array size\n");
        return 1;
    }

    int *arr, *d_arr, *temp;
    arr = (int *)malloc(n * sizeof(int));
    temp = (int *)malloc(n * sizeof(int));

    if (arr == NULL || temp == NULL)
    {
        printf("Memory allocation failed\n");
        return 1;
    }

    FILE *input_file = fopen("array1.txt", "r");
    if (input_file == NULL)
    {
        printf("Error opening file: array1.txt\n");
        free(arr);
        free(temp);
        return 1;
    }

    for (int i = 0; i < n; i++)
    {
        if (fscanf(input_file, "%d, ", &arr[i]) != 1)
        {
            printf("Error reading from file\n");
            fclose(input_file);
            free(arr);
            free(temp);
            return 1;
        }
    }
    hipMalloc(&d_arr, n * sizeof(int));
    hipMalloc(&temp, n * sizeof(int));

    int num_threads = THREADS_PER_BLOCK;
    int num_blocks = (n + num_threads - 1) / num_threads;

    clock_t total_time = 0;
    int num_trials = 5;

    for (int t = 0; t < num_trials; t++)
    {
        hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

        clock_t start = clock();

        mergeSort<<<num_blocks, num_threads, num_threads * sizeof(int)>>>(d_arr, temp, n);

        hipDeviceSynchronize();

        clock_t end = clock();
        total_time += (end - start);
        double elapsed_time = ((double)(end - start) / CLOCKS_PER_SEC) * 1000.0;

        // printf("Trial %d: Sorting time - %.2f milliseconds\n", t + 1, elapsed_time);
    }
    hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    mergeSortedBlocks(arr, n, THREADS_PER_BLOCK);
    double average_time_ms = ((double)total_time / num_trials) / CLOCKS_PER_SEC * 1000.0;
    printf("Average sorting time over %d trials: %.2f milliseconds\n", num_trials, average_time_ms);

    hipFree(d_arr);
    hipFree(temp);
    free(arr);

    return 0;
}
